
#include <hip/hip_runtime.h>
#include <math.h>
__device__
double tanhn(double x)
{
    double a;
    a=tanh(x);
    return a;
}
#include <math.h>
__device__
double ppow(double a , double b)
{
        double x;
        x=pow(a,b);
        return x;
}
#include <math.h>
__device__
double llog2(int a)
{
        int x,b,c;
        b=a;
     for(int i=0;i<10;i++)
     {
        b=b/2;
        if(b!=1)
        {
            x++;
        }
     }
        return x;
}
__device__
double atomicAddd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
__device__
double atomicMaxx(double* address, double val)
{
   unsigned long long int* address_as_i = (unsigned long long int*) address;
   unsigned long long int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
                        __double_as_longlong(fmax(val,
                                __longlong_as_double(assumed))));
    } while (assumed != old);
    return __longlong_as_double(old);
}
extern "C"
__global__ void non(){
int id = threadIdx.x;
};
__device__
void delay(int a)
{for(int i=0 ; i<a ;i++){printf("");}}
extern "C"
__global__ void test(double *in,double *out,double *w,double *b,int *pix,int *kernel){
int id = threadIdx.x;
int row=threadIdx.x / 5;
int col=threadIdx.x % 5;
int pixr = pix[0]/32;
int pixc = pix[0]%32;
int outkernel=kernel[0]%6;
out[(col+pixc)+(row+pixr)*36]+=in[0]*w[col+row*5];
//printf("id=%d ",id);
//printf("out[%d][%d] pix[%d][%d] w[%d][%d]\n",row+pixr,col+pixc,pixr,pixc,row,col);

};



//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////2222222222222222
extern "C"
__global__ void PartialCore(double *in,double *out,double *w,int *pix){
int id =  threadIdx.x;     //deviceinput deviceout2
int pixr = pix[0];
int xx=threadIdx.x + blockIdx.x*blockDim.x;
double m=0;
int n=0;
int z=0;
__syncthreads();
int i=blockIdx.x;
int ker = id  / 25;
int row=  id  % 25 /5;
int col=  id  % 25 %5;
__syncthreads();
__shared__ double Ns[5*5*6];
Ns[col+row*5+ker*5*5]=w[col+row*5+ker*5*5];
__syncthreads();
if(pixr%2==1)
{
    n=6;
}
__syncthreads();
//if(pix[0]<32)
{
out[(col+i+z)+(row)*36+(ker+n)*36*5]=0;
__syncthreads();
atomicAddd(&out[(col+i+z)+(row)*36+(ker+n)*36*5],in[i+z+pixr*32]*Ns[col+row*5+ker*5*5]);
__syncthreads();
}
};


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////2222222222222222222
extern "C"
__global__ void ACore(double *in,double *b,int *pix,double *tanout
,double *AM1,double *AM2,double *AM3,double *AM4,double *AM5,double *AM6)
{
int id = threadIdx.x;
int ker = blockIdx.x;
int pixr = pix[0];
int pixnum = pixr%6;
int n=0;
if(pixr%2==1)  n=6;

if(id<28)
  {
    if(pixr>4)
    {   int a=0;
        int col = id;
        if(pixnum==0) tanout[(col-a)+(0)*28+ker*28*28]=tanhn(AM2[col+ker*28]+b[ker]);

        if(pixnum==1) tanout[(col-a)+(1)*28+ker*28*28]=tanhn(AM3[col+ker*28]+b[ker]);

        if(pixnum==2) tanout[(col-a)+(0)*28+ker*28*28]=tanhn(AM4[col+ker*28]+b[ker]);

        if(pixnum==3) tanout[(col-a)+(1)*28+ker*28*28]=tanhn(AM5[col+ker*28]+b[ker]);

        if(pixnum==4) tanout[(col-a)+(0)*28+ker*28*28]=tanhn(AM6[col+ker*28]+b[ker]);

        if(pixnum==5) tanout[(col-a)+(1)*28+ker*28*28]=tanhn(AM1[col+ker*28]+b[ker]);
    }
 }


if(id>27)
{
    int a=4;
    int col=threadIdx.x % 28;
    {
           if(pixnum==0)
              {
                  if(id>27&&id<56)
                  AM1[col+ker*28]=in[col+a+4*36+(ker+n)*36*5];
                  if(id>55&&id<84)
                  AM3[col+ker*28]+=in[col+a+0*36+(ker+n)*36*5];
                  if(id>83&&id<112)
                  AM4[col+ker*28]+=in[col+a+1*36+(ker+n)*36*5];
                  if(id>111&&id<140)
                  AM5[col+ker*28]+=in[col+a+2*36+(ker+n)*36*5];
                  if(id>139&&id<168)
                  AM6[col+ker*28]+=in[col+a+3*36+(ker+n)*36*5];
              }
          if(pixnum==1)
              {
                  if(id>27&&id<56)
                  AM1[col+ker*28]+=in[col+a+3*36+(ker+n)*36*5];
                  if(id>55&&id<84)
                  AM2[col+ker*28]=in[col+a+4*36+(ker+n)*36*5];
                  if(id>83&&id<112)
                  AM4[col+ker*28]+=in[col+a+0*36+(ker+n)*36*5];
                  if(id>111&&id<140)
                 AM5[col+ker*28]+=in[col+a+1*36+(ker+n)*36*5];
                  if(id>139&&id<168)
                  AM6[col+ker*28]+=in[col+a+2*36+(ker+n)*36*5];
              }
          if(pixnum==2)
              {
                  if(id>27&&id<56)
                  AM1[col+ker*28]+=in[col+a+2*36+(ker+n)*36*5];
                  if(id>55&&id<84)
                  AM2[col+ker*28]+=in[col+a+3*36+(ker+n)*36*5];
                  if(id>83&&id<112)
                  AM3[col+ker*28]=in[col+a+4*36+(ker+n)*36*5];
                  if(id>111&&id<140)
                  AM5[col+ker*28]+=in[col+a+0*36+(ker+n)*36*5];
                  if(id>139&&id<168)
                  AM6[col+ker*28]+=in[col+a+1*36+(ker+n)*36*5];


              }
          if(pixnum==3)
              {
                  if(id>27&&id<56)
                  AM1[col+ker*28]+=in[col+a+1*36+(ker+n)*36*5];
                  if(id>55&&id<84)
                  AM2[col+ker*28]+=in[col+a+2*36+(ker+n)*36*5];
                  if(id>83&&id<112)
                  AM3[col+ker*28]+=in[col+a+3*36+(ker+n)*36*5];
                  if(id>111&&id<140)
                  AM4[col+ker*28]=in[col+a+4*36+(ker+n)*36*5];
                  if(id>139&&id<168)
                  AM6[col+ker*28]+=in[col+a+0*36+(ker+n)*36*5];
              }

          if(pixnum==4)
              {
                  if(id>27&&id<56)
                  AM1[col+ker*28]+=in[col+a+0*36+(ker+n)*36*5];
                  if(id>55&&id<84)
                  AM2[col+ker*28]+=in[col+a+1*36+(ker+n)*36*5];
                  if(id>83&&id<112)
                  AM3[col+ker*28]+=in[col+a+2*36+(ker+n)*36*5];
                  if(id>111&&id<140)
                  AM4[col+ker*28]+=in[col+a+3*36+(ker+n)*36*5];
                  if(id>139&&id<168)
                  AM5[col+ker*28]=in[col+a+4*36+(ker+n)*36*5];

              }
          if(pixnum==5)
              {

                 if(id>27&&id<56)
                 AM2[col+ker*28]+=in[col+a+0*36+(ker+n)*36*5];
                 if(id>55&&id<84)
                 AM3[col+ker*28]+=in[col+a+1*36+(ker+n)*36*5];
                 if(id>83&&id<112)
                 AM4[col+ker*28]+=in[col+a+2*36+(ker+n)*36*5];
                 if(id>111&&id<140)
                 AM5[col+ker*28]+=in[col+a+3*36+(ker+n)*36*5];
                 if(id>139&&id<168)
                 AM6[col+ker*28]=in[col+a+4*36+(ker+n)*36*5];

              }
    }

}


};


 ////////////////////////////////////////////////////////////////////////////////////////////////////////////////
 extern "C"
 __global__ void poolcore(double *pool,double *in,int *pix){

    int id = threadIdx.x;
    int ker = blockIdx.x;
    int col = id % 28;
    int Crow;
    int indexx=1;
    Crow = pix[0]-5;
    if(Crow%2==1) indexx=0;

    if(Crow>=0)
    {
        atomicMaxx(&pool[(col/2)+(Crow/2)*14+ker*14*14],in[col+(indexx)*28+ker*28*28]);
    }
 };












////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern "C"
 __global__ void Pool2Core(double *pool,double *in,int *index,int *pix){
    int xx = threadIdx.x + blockIdx.x*blockDim.x;
    int id = threadIdx.x;
    int ker = blockIdx.x;
    int col = threadIdx.x %10;

    int Crow;
    Crow = pix[0]-5;
    int indexxx = index[0];
    int inndex = 1;
    if(Crow%2==1) inndex=0;
    __syncthreads();


        if(Crow>=0)
        {


            atomicMaxx(&pool[(col/2)*16+(Crow/2)*5*16+ker]   ,in[col+(inndex)*10+ker*10*10]);
            __syncthreads();



        }



 };
/////////222222222222222222222222222222222222222222222222222222222
extern "C"
__global__ void Partial2Core(double *in,double *out,double *w,int *pix){
int id = blockIdx.x * blockDim.x + threadIdx.x;    //deviceinputrow2 deviceout2_2
int pixr = pix[0];

int ker = threadIdx.x / 25;
int j =blockIdx.x ;// /6
int row = threadIdx.x %25 / 5;
int col = threadIdx.x %25 % 5;

int n =0;
double ac;
int xx=threadIdx.x + blockIdx.x*blockDim.x;


if(pixr%2==1) n=16;

{


                out[(col+j)+(row)*18+(ker+n)*18*5]=0;
                __syncthreads();


                for(int iiker=0;iiker<6;iiker++)

                {

                    atomicAddd(&out[(col+j)+(row)*18+(ker+n)*18*5] , in[j+pixr*14+iiker*14*14]*w[col+row*5+iiker*5*5+ker*5*5*6]);
                    __syncthreads();
                }


}
};
////////////////////////////////////////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void A2Core(double *in,double *b,int *pix,double *tanout,double *AM1,double *AM2,double *AM3,
double *AM4,double *AM5,double *AM6)//
{
int xx = threadIdx.x + blockIdx.x*blockDim.x;
int id = threadIdx.x;
int ker = blockIdx.x;
int n =0;
int z =0;


int pixr = pix[0];
int pixnum = pixr%6;
if(pix[0]%2==1)n=16;
{

    if(id<10)
    {
        int col=threadIdx.x;
        int a=0;
        if(pixr>4)
            {
                    if(pixnum==0)
                        {

                          tanout[(col-a)+(0)*10+ker*10*10]=tanhn(AM2[col+ker*10]+b[ker]);


                        }
                    if(pixnum==1)
                        {

                             tanout[(col-a)+(1)*10+ker*10*10]=tanhn(AM3[col+ker*10]+b[ker]);



                        }
                    if(pixnum==2)
                        {

                            tanout[(col-a)+(0)*10+ker*10*10]=tanhn(AM4[col+ker*10]+b[ker]);

                        }
                    if(pixnum==3)
                        {

                            tanout[(col-a)+(1)*10+ker*10*10]=tanhn(AM5[col+ker*10]+b[ker]);



                        }

                    if(pixnum==4)
                        {

                            tanout[(col-a)+(0)*10+ker*10*10]=tanhn(AM6[col+ker*10]+b[ker]);


                        }
                    if(pixnum==5)
                        {

                           tanout[(col-a)+(1)*10+ker*10*10]=tanhn(AM1[col+ker*10]+b[ker]);


                        }

            }
    }
}


{
    if(id>9)
    {
        int col= threadIdx.x % 10;
        int a=4;


        {
            if(pixnum==0)
            {
		            if(id>9&&id<20)
            AM1[col+ker*10]=in[col+a+4*18+(ker+n)*18*5];    //+ker*18*10     +ker*18*5
                  if(id>19&&id<30)
            AM3[col+ker*10]+=in[col+a+0*18+(ker+n)*18*5];
                  if(id>29&&id<40)
            AM4[col+ker*10]+=in[col+a+1*18+(ker+n)*18*5];
                  if(id>39&&id<50)
            AM5[col+ker*10]+=in[col+a+2*18+(ker+n)*18*5];
                  if(id>49&&id<60)
            AM6[col+ker*10]+=in[col+a+3*18+(ker+n)*18*5];
            //__syncthreads();

           // AM2[col+ker*10]=0;



            }
        if(pixnum==1)
            {

		  if(id>9&&id<20)
            AM1[col+ker*10]+=in[col+a+3*18+(ker+n)*18*5];
                  if(id>19&&id<30)
            AM2[col+ker*10]=in[col+a+4*18+(ker+n)*18*5];
                  if(id>29&&id<40)
            AM4[col+ker*10]+=in[col+a+0*18+(ker+n)*18*5];
                  if(id>39&&id<50)
            AM5[col+ker*10]+=in[col+a+1*18+(ker+n)*18*5];
                  if(id>49&&id<60)
            AM6[col+ker*10]+=in[col+a+2*18+(ker+n)*18*5];


            }
        if(pixnum==2)
            {
		  if(id>9&&id<20)
            AM1[col+ker*10]+=in[col+a+2*18+(ker+n)*18*5];
                  if(id>19&&id<30)
            AM2[col+ker*10]+=in[col+a+3*18+(ker+n)*18*5];
                  if(id>29&&id<40)
            AM3[col+ker*10]=in[col+a+4*18+(ker+n)*18*5];
                  if(id>39&&id<50)
            AM5[col+ker*10]+=in[col+a+0*18+(ker+n)*18*5];
                  if(id>49&&id<60)
            AM6[col+ker*10]+=in[col+a+1*18+(ker+n)*18*5];




            }
        if(pixnum==3)
            {
		  if(id>9&&id<20)
            AM1[col+ker*10]+=in[col+a+1*18+(ker+n)*18*5];
                  if(id>19&&id<30)
            AM2[col+ker*10]+=in[col+a+2*18+(ker+n)*18*5];
                  if(id>29&&id<40)
            AM3[col+ker*10]+=in[col+a+3*18+(ker+n)*18*5];
                  if(id>39&&id<50)
            AM4[col+ker*10]=in[col+a+4*18+(ker+n)*18*5];
                  if(id>49&&id<60)
            AM6[col+ker*10]+=in[col+a+0*18+(ker+n)*18*5];




            }
            ///////////////////////////////////////////////////////////////////
        if(pixnum==4)
            {

		  if(id>9&&id<20)
            AM1[col+ker*10]+=in[col+a+0*18+(ker+n)*18*5];
                  if(id>19&&id<30)
            AM2[col+ker*10]+=in[col+a+1*18+(ker+n)*18*5];
                  if(id>29&&id<40)
            AM3[col+ker*10]+=in[col+a+2*18+(ker+n)*18*5];
                  if(id>39&&id<50)
            AM4[col+ker*10]+=in[col+a+3*18+(ker+n)*18*5];
                  if(id>49&&id<60)
            AM5[col+ker*10]=in[col+a+4*18+(ker+n)*18*5];







            }
        if(pixnum==5)
            {
          if(id>9&&id<20)
           AM2[col+ker*10]+=in[col+a+0*18+(ker+n)*18*5];
                  if(id>19&&id<30)
           AM3[col+ker*10]+=in[col+a+1*18+(ker+n)*18*5];
                  if(id>29&&id<40)
           AM4[col+ker*10]+=in[col+a+2*18+(ker+n)*18*5];
                  if(id>39&&id<50)
           AM5[col+ker*10]+=in[col+a+3*18+(ker+n)*18*5];
                  if(id>49&&id<60)
           AM6[col+ker*10]=in[col+a+4*18+(ker+n)*18*5];





            }
            //__syncthreads();
        }

    }
}

};

/////////////////////////////////////////////////////////////////////////////
extern "C"
__global__ void FC(int *par,double *w, double *s, double *in,double *b){
int inSize=par[0],outSize=par[1];
int id = threadIdx.x;
int row = id / outSize;
int col = id % outSize;
double tol;
if (id <outSize)
{
  tol=0;
  for(int j=0; j<inSize; j++)

  {
   tol+=in[j]*w[id+j*outSize];

  }
    __syncthreads();


    s[id]=tanh(tol+b[id]);
    __syncthreads();
}
};

extern "C"
__global__ void FCL3(int *par,double *w, double *s, double *in,double *b){
int inSize=par[0],outSize=par[1];
double E=2.7182818284590452354;
__shared__ double exp[10];
int id = threadIdx.x + blockIdx.x*blockDim.x;
int row = id / outSize;
int col = id % outSize;
double tol;
if (id <outSize)
{
  exp[id]=0;
  tol=0;
  for(int j=0; j<inSize; j++)
  {
   tol+=in[j]*w[id+j*outSize];
  }
 __syncthreads();
 tol=tol+b[id];
 exp[id]+=ppow(E,tol);
 exp[0]=exp[0]+exp[1]+exp[2]+exp[3]+exp[4]+exp[5]+exp[6]+exp[7]+exp[8]+exp[9];
 __syncthreads();

 s[id]=ppow(E,tol)/exp[0];

}
};
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

