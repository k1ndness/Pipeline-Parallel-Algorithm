
#include <hip/hip_runtime.h>
#include <math.h>
__device__
double tanhn(double x)
{
    double a;
    a=tanh(x);
    return a;
}
#include <math.h>
__device__
double ppow(double a , double b)
{
        double x;
        x=pow(a,b);
        return x;
}
extern "C"
__global__ void add(int *n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n[0])
    {
        sum[i] = a[i] + b[i];
    }

};
const int XXX = 64*64;
extern "C"
__global__ void PARSAD(int *par,double *s, double *in, double*w, double*b){
 int width=par[0],height=par[1],channeltime=par[2],ichan=par[3],kernelSize=par[4];//32,32,6,1,5
 int id = threadIdx.x + blockIdx.x*blockDim.x;                                    //14,14,16,6,5

 if (id <width*height*channeltime)
 {
 int channel = id/(width*height);
 int row = (id %(width*height))/ width;
 int col = (id %(width*height))% width;
 int x=col-4,y=row-4;
 int z=width-kernelSize+1;
 int zh=height-kernelSize+1;
//__shared__ double Ns[XXX];
//Ns[col+row*width]=in[col+row*width];
//__syncthreads();
   for (int ch = 0; ch < ichan; ch++) {
     for (int i = 0; i < kernelSize; i++)
     {
      for (int j = 0; j < kernelSize; j++)
      {
          if(row+i>3&&col+j>3&&row+i<height&&col+j<width)
          {
          s[(x+j)+(y+ i)*z+channel*z*zh]+=in[col+row*width+ch*width*width]
          *w[j+i*kernelSize+ch*kernelSize*kernelSize+channel*ichan*kernelSize*kernelSize];
          }
          __syncthreads();

      }
     }
       if(row>3&&col>3&&ch==ichan-1)
       s[(x)+(y)*z+channel*z*zh]=tanhn(s[(x)+(y)*z+channel*z*zh]+b[channel]);
   }
 }
};
extern "C"
__global__ void Conv1(int *par,double *s, double *in, double*w, double*b){

 int id = blockIdx.x * blockDim.x + threadIdx.x;
int insizeW=par[0],insizeH=par[1],outchannel=par[2],inchannel=par[3],kernelsize=par[4],stride=1;
  int outsizeW=insizeW-kernelsize+1;
  int outsizeH=insizeH-kernelsize+1;
    int channel, row, col;
    int ch, i, j;

    if (id < outsizeW*outsizeH*outchannel) {
        channel = id / (outsizeW * outsizeH);
        row = (id % (outsizeW * outsizeH)) / outsizeW;
        col = (id % (outsizeW * outsizeH)) % outsizeW;

        s[channel*outsizeW*outsizeH+row*outsizeW+col] = 0.0;
        for (ch = 0; ch < inchannel; ch++) {
        for (i = 0; i < kernelsize; i++) {
            for (j = 0; j < kernelsize; j++) {
                   s[channel*outsizeW*outsizeH+row*outsizeW+col] +=w[channel*inchannel*kernelsize*kernelsize+ch*kernelsize*kernelsize+i*kernelsize+j]
                   *in[ch*insizeW*insizeH+(i+row)*insizeW+(j+col)];
                    __syncthreads();
                }
            }
        }
          s[channel*outsizeW*outsizeH+row*outsizeW+col] = tanhn(s[channel*outsizeW*outsizeH+row*outsizeW+col]+b[channel]);
    }

};
extern "C"
__global__ void PARSADAC(int *par,double *s, double*b){
 int width=par[0]-4,height=par[1]-4,channeltime=par[2],ichan=par[3],kernelSize=par[4];//32,32,6,1,5
 int id = threadIdx.x + blockIdx.x*blockDim.x;                                    //14,14,16,6,5

 if (id <width*height*channeltime)
 {

 int channel = id/(width*height);
 int row = (id %(width*height))/ width;
 int col = (id %(width*height))% width;
 int x=col-4,y=row-4;
 int z=width-kernelSize+1;
 int zh=height-kernelSize+1;

       s[(x)+(y)*z+channel*z*zh]=tanhn(s[(x)+(y)*z+channel*z*zh]+b[channel]);

 }
};










extern "C"
__global__ void pool1(int *par,double *pool,double *s){
       int id =threadIdx.x + blockIdx.x*blockDim.x;    //14,14,6,28
       int width=par[0],height=par[1],channel,channeltime=par[2],z=par[3];
       int row,col;
       double max,tmp;
       if (id <width * height*channeltime)
        {
        channel = id/(width*height);
        row = (id %(width*height))/ width;
        col = (id %(width*height))% width;
        max=-256.0;
           for(int k = 0 ; k < 2 ; k++)
           {
              for (int l = 0 ; l < 2 ; l++)
              {
                tmp=s[(col*2+l)+(row*2*z+k*z)+channel*z*z];
                if(max < tmp)
                {
                max=tmp;
                }
              }
              if(width==14)
              pool[col+row*width+channel*width*height]=max;
              if(width==5)
              pool[col*16+row*5*16+channel]=max;
           }

        }
};
extern "C"
__global__ void PARSAD2(int *par,double *s, double *in, double*w, double*b){
 int width=14,height=14,channeltime=16,ichan=6,kernelSize=5;//32,32,6,1,5
 int id = threadIdx.x + blockIdx.x*blockDim.x;                                    //14,14,16,6,5

 if (id <width*height*channeltime)
 {

 int channel = id/(width*height);
 int row = (id %(width*height))/ width;
 int col = (id %(width*height))% width;
 int z=width-kernelSize+1;
   for (int ch = 0; ch < ichan; ch++) {
     for (int i = 0; i < kernelSize; i++)
     {
      for (int j = 0; j < kernelSize; j++)
      {
          if(row+i>3&&col+j>3&&row+i<width&&col+j<width)
          s[(col+j-4)+(row+ i-4)*z+channel*z*z]+=in[col+row*width+ch*width*width]*
          w[j+i*kernelSize+ch*kernelSize*kernelSize+channel*ichan*kernelSize*kernelSize];
          __syncthreads();
      }
     }
      if(row>3&&col>3&&ch==ichan-1)
      s[(col-4)+(row-4)*z+channel*z*z]=tanhn(s[(col-4)+(row-4)*z+channel*z*z]+b[channel]);

   }


 }



};
extern "C"
__global__ void FC(int *par,double *w, double *s, double *in,double *b){
int inSize=par[0],outSize=par[1];
int id = threadIdx.x + blockIdx.x*blockDim.x;
int row = id / outSize;
int col = id % outSize;
double tol;
if (id <outSize)
{
  tol=0;
  for(int j=0; j<inSize; j++)
  {
   tol+=in[j]*w[id+j*outSize];
  }
 __syncthreads();

 s[id]=tanh(tol+b[id]);

}
};

extern "C"
__global__ void FCL3(int *par,double *w, double *s, double *in,double *b){
int inSize=par[0],outSize=par[1];
double E=2.7182818284590452354;
__shared__ double exp[10];
int id = threadIdx.x + blockIdx.x*blockDim.x;
int row = id / outSize;
int col = id % outSize;
double tol;
if (id <outSize)
{
  exp[id]=0;
  tol=0;
  for(int j=0; j<inSize; j++)
  {
   tol+=in[j]*w[id+j*outSize];
  }
 __syncthreads();
 tol=tol+b[id];
 exp[id]+=ppow(E,tol);
 exp[0]=exp[0]+exp[1]+exp[2]+exp[3]+exp[4]+exp[5]+exp[6]+exp[7]+exp[8]+exp[9];
 __syncthreads();

 s[id]=ppow(E,tol)/exp[0];

}
};










